#include "hip/hip_runtime.h"
/*
 * This software contains source code provided by NVIDIA Corporation
 *
 * According to the Nvidia EULA (compute 5.5 version)
 * http://developer.download.nvidia.com/compute/cuda/5_5/rel/docs/EULA.pdf
 *
 * Chapter 2. NVIDIA CORPORATION CUDA SAMPLES END USER LICENSE AGREEMENT
 * 2.1.1. Source Code
 * Developer shall have the right to modify and create derivative works with the Source
 * Code. Developer shall own any derivative works ("Derivatives") it creates to the Source
 * Code, provided that Developer uses the Materials in accordance with the terms and
 * conditions of this Agreement. Developer may distribute the Derivatives, provided that
 * all NVIDIA copyright notices and trademarks are propagated and used properly and
 * the Derivatives include the following statement: “This software contains source code
 * provided by NVIDIA Corporation.”
 */

/*
 * This application demonstrates how to use the CUDA API to use multiple GPUs,
 * with an emphasis on simple illustration of the techniques (not on performance).
 *
 * Note that in order to detect multiple GPUs in your system you have to disable
 * SLI in the nvidia control panel. Otherwise only one GPU is visible to the
 * application. On the other side, you can still extend your desktop to screens
 * attached to both GPUs.
 */

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cupti.h>

#include "papi.h"
#include "papi_test.h"

typedef struct {
    int dataN;
    float *h_Data;
    float *h_Sum;
    float *d_Data,*d_Sum;
    float *h_Sum_from_device;
    hipStream_t stream;
} TGPUplan;

const int MAX_GPU_COUNT = 32;
const int DATA_N = 48576 * 32;
const int MAX_NUM_EVENTS = 32;

__global__ static void reduceKernel(float *d_Result, float *d_Input, int N)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int threadN = gridDim.x * blockDim.x;
    float sum = 0;

    for (int pos = tid; pos < N; pos += threadN) {
        sum += d_Input[pos];
    }

    d_Result[tid] = sum;
}

int main(int argc, char **argv)
{
    TGPUplan plan[MAX_GPU_COUNT];
    float h_SumGPU[MAX_GPU_COUNT];
    float sumGPU;
    double sumCPU, diff;
    int i, j, gpuBase, GPU_N;

    const int BLOCK_N = 32;
    const int THREAD_N = 256;
    const int ACCUM_N = BLOCK_N * THREAD_N;

    int quiet = tests_quiet(argc, argv);

    hipCtx_t ctx[MAX_GPU_COUNT];
    hipCtx_t poppedCtx;
    hipError_t cu_errno = hipSuccess;

    printf("Starting cupti_single_thread_monitoring\n");

    int papi_errno = PAPI_library_init(PAPI_VER_CURRENT);
    if (papi_errno != PAPI_VER_CURRENT) {
        fprintf(stderr, "PAPI_library_init failed\n");
        test_fail(__FILE__, __LINE__, "PAPI_library_init failed.", 0);
    }

    printf("PAPI version: %d.%d.%d\n", PAPI_VERSION_MAJOR(PAPI_VERSION),
                                       PAPI_VERSION_MINOR(PAPI_VERSION),
                                       PAPI_VERSION_REVISION(PAPI_VERSION));

    int cc_major = 0, cc_minor = 0;
    int runtimeVersion = 0, driverVersion = 0;
    char deviceName[64];
    hipDevice_t device[MAX_GPU_COUNT];
    hipError_t cuda_errno = hipSuccess;

    cuda_errno = hipGetDeviceCount(&GPU_N);
    if (cuda_errno != hipSuccess) {
        test_fail(__FILE__, __LINE__, "hipGetDeviceCount failed.", 0);
    }

    if (GPU_N > MAX_GPU_COUNT) {
        GPU_N = MAX_GPU_COUNT;
    }

    printf("CUDA-capable device count: %i\n", GPU_N);
    for (i = 0; i < GPU_N; i++) {
        cu_errno = hipDeviceGet(&device[i], i);
        if (cu_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipDeviceGet failed.", 0);
        }

        cu_errno = hipDeviceGetName(deviceName, 64, device[i]);
        if (cu_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipDeviceGetName failed.", 0);
        }

        cu_errno = hipDeviceGetAttribute(&cc_major,
                                        hipDeviceAttributeComputeCapabilityMajor,
                                        device[i]);
        if (cu_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipDeviceGetAttribute failed.", 0);
        }

        cu_errno = hipDeviceGetAttribute(&cc_minor,
                                        hipDeviceAttributeComputeCapabilityMinor,
                                        device[i]);
        if (cu_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipDeviceGetAttribute failed.", 0);
        }

        hipRuntimeGetVersion(&runtimeVersion);
        hipDriverGetVersion(&driverVersion);

        printf("CUDA Device %d: %s : computeCapability %d.%d runtimeVersion %d.%d driverVersion %d.%d\n",
                i, deviceName, cc_major, cc_minor, runtimeVersion/1000, (runtimeVersion % 100) / 10,
                driverVersion / 1000, (driverVersion % 100) / 10);

        if (cc_major < 2) {
            printf("CUDA Device %d compute capability is too low... will not add any more GPUs\n", i);
            GPU_N = i;
            break;
        }
    }

    uint32_t cupti_linked_version;
    cuptiGetVersion(&cupti_linked_version);
    printf("CUPTI version: Compiled against version %d; Linked against version %d\n",
           CUPTI_API_VERSION, cupti_linked_version );

    for (i = 0; i < GPU_N; i++) {
        cu_errno = hipCtxCreate(&ctx[i], 0, device[i]);
        if (cu_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipCtxCreate failed.", 0);
        }

        cu_errno = hipCtxPopCurrent(&poppedCtx);
        if (cu_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipCtxPopCurrent failed.", 0);
        }
    }

    printf("Generating input data...\n");

    for (i = 0; i < GPU_N; i++)
        plan[i].dataN = DATA_N / GPU_N;

    for (i = 0; i < DATA_N % GPU_N; i++)
        plan[i].dataN++;

    gpuBase = 0;
    for (i = 0; i < GPU_N; i++) {
        plan[i].h_Sum = h_SumGPU + i;
        gpuBase += plan[i].dataN;
    }

    for (i = 0; i < GPU_N; i++) {
        cu_errno = hipCtxPushCurrent(ctx[i]);
        if (cu_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipCtxPushCurrent failed.", 0);
        }

        cuda_errno = hipStreamCreate(&plan[i].stream);
        if (cuda_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipStreamCreate failed.", 0);
        }

        cuda_errno = hipMalloc((void **) &plan[i].d_Data, plan[i].dataN * sizeof(float));
        if (cuda_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipMalloc failed." , 0);
        }

        cuda_errno = hipMalloc((void **) &plan[i].d_Sum, ACCUM_N * sizeof(float));
        if (cuda_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipMalloc failed.", 0);
        }

        cuda_errno = hipHostMalloc((void **) &plan[i].h_Sum_from_device, ACCUM_N * sizeof(float));
        if (cuda_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipHostMalloc failed.", 0);
        }

        cuda_errno = hipHostMalloc((void **) &plan[i].h_Data, plan[i].dataN * sizeof(float));
        if (cuda_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipHostMalloc failed.", 0);
        }

        for (j = 0; j < plan[i].dataN; j++) {
            plan[i].h_Data[j] = (float) rand() / (float) RAND_MAX;
        }

        cu_errno = hipCtxPopCurrent(&poppedCtx);
        if (cu_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipCtxPopCurrent failed.", 0);
        }
    }

    if (!quiet) {
        printf("Setup PAPI counters internally (PAPI)\n");
    }

    int EventSet = PAPI_NULL;
    int NUM_EVENTS = MAX_GPU_COUNT * MAX_NUM_EVENTS;
    long long values[NUM_EVENTS];
    int eventCount;
    int cid = -1;
    int ee;

    int k = PAPI_num_components();
    for (i = 0; i < k && cid < 0; i++) {
        PAPI_component_info_t *aComponent =
            (PAPI_component_info_t*) PAPI_get_component_info(i);
        if (aComponent == NULL) {
            fprintf(stderr,  "PAPI_get_component_info(%i) failed, "
                    "returned NULL. %i components reported.\n", i, k);
            test_fail(__FILE__, __LINE__, "PAPI_get_component_info failed.", 0);
        }

        if (strcmp("cuda", aComponent->name) == 0) {
            cid = i;
        }
    }

    if (cid < 0) {
        fprintf(stderr, "Failed to find cuda component among %i "
                "reported components.\n", k);
        test_fail(__FILE__, __LINE__, "", 0);
    }

    if (!quiet) {
        printf("Found CUDA Component at id %d\n", cid);
    }

    papi_errno = PAPI_create_eventset(&EventSet);
    if (papi_errno != PAPI_OK) {
        test_fail(__FILE__, __LINE__, "PAPI_create_eventset failed.", papi_errno);
    }

    papi_errno = PAPI_assign_eventset_component(EventSet, cid);
    if (papi_errno != PAPI_OK) {
        test_fail(__FILE__, __LINE__, "PAPI_assign_eventset_component failed.",
                  papi_errno);
    }

    int numEventNames = 2;

    char const *EventNames[2];
    if (cc_major < 7 || (cc_major == 7 && cc_minor == 0)) {
        EventNames[0] = "cuda:::event:active_cycles_pm";
        EventNames[1] = "cuda:::event:active_warps_pm";
    } else {
        EventNames[0] = "cuda:::dram__bytes_read.sum";
        EventNames[1] = "cuda:::sm__warps_launched.sum";
    }

    // Add events at a GPU specific level ... eg cuda:::device:2:elapsed_cycles_sm
    // UNLIKE CUpti_11, we must change the contexts to the appropriate device to
    // add events to inform PAPI of the context that will run the kernels.

    // Save current context, will restore after adding events.
    hipCtx_t userContext;
    cu_errno = hipCtxGetCurrent(&userContext);
    if (cu_errno != hipSuccess) {
        test_fail(__FILE__, __LINE__, "hipCtxGetCurrent failed.", 0);
    }

    char *EventName[NUM_EVENTS];
    char tmpEventName[64];
    eventCount = 0;
    for (i = 0; i < GPU_N; i++) {
        cu_errno = hipCtxSetCurrent(ctx[i]);
        if (cu_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipCtxSetCurrent failed.", 0);
        }

        for (ee = 0; ee < numEventNames; ee++) {
            snprintf(tmpEventName, 64, "%s:device=%d\0", EventNames[ee], i);
            papi_errno = PAPI_add_named_event(EventSet, tmpEventName);
            if (papi_errno == PAPI_OK) {
                if (!quiet) {
                    printf("Add event success: '%s' GPU %i\n", tmpEventName, i);
                }
                EventName[eventCount] = (char *)calloc(64, sizeof(char));
                snprintf(EventName[eventCount], 64, "%s", tmpEventName);
                eventCount++;
            } else {
                if (!quiet) {
                    printf("Add event failure: '%s' GPU %i error=%s\n",
                           tmpEventName, i, PAPI_strerror(papi_errno));
                }
            }
        }
    }

    cu_errno = hipCtxSetCurrent(userContext);
    if (cu_errno != hipSuccess) {
        test_fail(__FILE__, __LINE__, "hipCtxSetCurrent failed.", 0);
    }

    papi_errno = PAPI_start(EventSet);
    if( papi_errno != PAPI_OK ) {
        test_fail(__FILE__, __LINE__, "PAPI_start failed.", papi_errno);
    }

    if (!quiet) {
        printf("Computing with %d GPUs...\n", GPU_N);
    }

    for (i = 0; i < GPU_N; i++) {
        cu_errno = hipCtxPushCurrent(ctx[i]);
        if (cu_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipCtxPushCurrent failed.", 0);
        }

        cuda_errno = hipMemcpyAsync(plan[i].d_Data, plan[i].h_Data,
                                     plan[i].dataN * sizeof( float ),
                                     hipMemcpyHostToDevice, plan[i].stream );

        reduceKernel<<<BLOCK_N, THREAD_N, 0, plan[i].stream>>>(plan[i].d_Sum, plan[i].d_Data, plan[i].dataN);

        if (hipGetLastError() != hipSuccess) {
            printf( "reduceKernel() execution failed (GPU %d).\n", i );
            test_fail(__FILE__, __LINE__, "reduceKernel failed.", 0);
        }

        cuda_errno = hipMemcpyAsync(plan[i].h_Sum_from_device, plan[i].d_Sum, ACCUM_N * sizeof( float ),
                                     hipMemcpyDeviceToHost, plan[i].stream );

        cu_errno = hipCtxPopCurrent(&(ctx[i]));
        if (cu_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipCtxPopCurrent failed.", 0);
        }
    }

    if (!quiet) {
        printf("Process GPU results on %d GPUs...\n", GPU_N);
    }

    for (i = 0; i < GPU_N; i++) {
        float sum;
        cu_errno = hipCtxPushCurrent(ctx[i]);
        if (cu_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipCtxPushCurrent failed.", 0);
        }

        hipStreamSynchronize(plan[i].stream);

        sum = 0;
        for (j = 0; j < ACCUM_N; j++) {
            sum += plan[i].h_Sum_from_device[j];
        }

        *(plan[i].h_Sum) = (float) sum;

        cu_errno = hipCtxPopCurrent(&ctx[i]);
        if (cu_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipCtxPopCurrent failed.", 0);
        }
    }

    for (i = 0; i < GPU_N; i++) {
        cu_errno = hipCtxPushCurrent(ctx[i]);
        if (cu_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "cuCtxPushCurrnet failed.", 0);
        }

        cu_errno = hipCtxSynchronize();
        if (cu_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipCtxSynchronize failed.", 0);
        }

        cu_errno = hipCtxPopCurrent(&ctx[i]);
        if (cu_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipCtxPopCurrent failed.", 0);
        }
    }

    papi_errno = PAPI_stop(EventSet, values);
    if (papi_errno != PAPI_OK ) {
        test_fail(__FILE__, __LINE__, "PAPI_stop failed.", papi_errno);
    }

    if (!quiet) {
        for (i = 0; i < eventCount; i++)
            printf("PAPI counterValue %12lld \t\t --> %s \n", values[i], EventName[i]);
    }

    int warp_size[GPU_N];
    int events_per_dev = eventCount / GPU_N;
    for (i = 0; i < GPU_N; i++) {
        hipDeviceGetAttribute(&warp_size[i], hipDeviceAttributeWarpSize, device[i]);
        if (values[(i * events_per_dev) + 1] != ((BLOCK_N * THREAD_N) / warp_size[i])) {
            test_fail(__FILE__, __LINE__, "", 0);
        }
    }

    papi_errno = PAPI_cleanup_eventset(EventSet);
    if (papi_errno != PAPI_OK) {
        test_fail(__FILE__, __LINE__, "PAPI_cleanup_eventset failed.", papi_errno);
    }

    papi_errno = PAPI_destroy_eventset(&EventSet);
    if (papi_errno != PAPI_OK) {
        test_fail(__FILE__, __LINE__, "PAPI_destroy_eventset failed.", papi_errno);
    }

    PAPI_shutdown();

    sumGPU = 0;
    for (i = 0; i < GPU_N; i++) {
        sumGPU += h_SumGPU[i];
    }

    if (!quiet) {
        printf("Computing the same result with Host CPU...\n");
    }

    sumCPU = 0;
    for (i = 0; i < GPU_N; i++) {
        for (j = 0; j < plan[i].dataN; j++) {
            sumCPU += plan[i].h_Data[j];
        }
    }

    if (!quiet) {
        printf("Comparing GPU and Host CPU results...\n");
    }

    diff = fabs(sumCPU - sumGPU) / fabs(sumCPU);

    if (!quiet) {
        printf("  GPU sum: %f\n  CPU sum: %f\n", sumGPU, sumCPU);
        printf("  Relative difference: %E \n", diff);
    }

    for (i = 0; i < GPU_N; i++) {
        hipHostFree(plan[i].h_Sum_from_device);
        hipHostFree(plan[i].h_Data);
        hipFree(plan[i].d_Sum);
        hipFree(plan[i].d_Data);
        hipStreamDestroy(plan[i].stream);
    }

    if (diff < 1e-5) {
        test_pass(__FILE__);
    } else {
        test_fail(__FILE__, __LINE__, "", 0);
    }

    return 0;
}
