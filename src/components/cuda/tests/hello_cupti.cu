#include "hip/hip_runtime.h"
/****************************/
/* THIS IS OPEN SOURCE CODE */
/****************************/

/**
 * @file    hello_cupti.cu
 * @author  Heike Jagode
 *          jagode@eecs.utk.edu
 * Mods:    Giuseppe Congiu
 *          gcongiu@icl.utk.edu
 * test case for Example component
 *
 *
 * @brief
 *  This file is a very simple HelloWorld C example which serves (together
 *  with its Makefile) as a guideline on how to add tests to components.
 *  The papi configure and papi Makefile will take care of the compilation
 *  of the component tests (if all tests are added to a directory named
 *  'tests' in the specific component dir).
 *  See components/README for more details.
 *
 *  The string "Hello Cupti!" is mangled and then restored.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "papi.h"
#include "papi_test.h"

#define NUM_EVENTS 1

// Prototypes
__global__ void helloCupti(char*);

static int
is_compute_capability_pre_cupti11()
{
    int cc_major, cc_minor;
    hipDevice_t device;
    hipError_t cuErr;

    cuErr = hipInit(0);
    if (cuErr != hipSuccess) {
        const char *errString = NULL;
        hipDrvGetErrorString(cuErr, &errString);
        fprintf(stderr, "%s:%s:%i hipInit hipError_t='%s'.\n",
                __FILE__, __func__, __LINE__, errString);
        test_fail(__FILE__, __LINE__, "hipInit failed", 0);
    }


    cuErr = hipDeviceGet(&device, 0);
    if (cuErr != hipSuccess) {
        const char *errString = NULL;
        hipDrvGetErrorString(cuErr, &errString);
        fprintf(stderr, "%s:%s:%i cuGetDevice hipError_t='%s'.\n",
                __FILE__, __func__, __LINE__, errString);
        test_fail(__FILE__, __LINE__, "cuGetDevice failed", 0);
    }

    cuErr = hipDeviceGetAttribute(&cc_major, hipDeviceAttributeComputeCapabilityMajor, device);
    if (cuErr != hipSuccess) {
        const char *errString = NULL;
        hipDrvGetErrorString(cuErr, &errString);
        fprintf(stderr, "%s:%s:%i cuGetAttribute hipError_t='%s'.\n",
                __FILE__, __func__, __LINE__, errString);
        test_fail(__FILE__, __LINE__, "cuGetAttribute failed", 0);
    }

    cuErr = hipDeviceGetAttribute(&cc_minor, hipDeviceAttributeComputeCapabilityMinor, device);
    if (cuErr != hipSuccess) {
        const char *errString = NULL;
        hipDrvGetErrorString(cuErr, &errString);
        fprintf(stderr, "%s:%s:%i cuGetAttribute hipError_t='%s'.\n",
                __FILE__, __func__, __LINE__, errString);
        test_fail(__FILE__, __LINE__, "cuGetAttribute failed", 0);
    }

    return (cc_major < 7 || (cc_major == 7 && cc_minor == 0));
}

// Host function
int main(int argc, char** argv)
{
    int retval, i;
    int EventSet = PAPI_NULL;
    long long values[NUM_EVENTS];
    int events[NUM_EVENTS];
    int eventCount = 0;
    int quiet;

    const char *EventName[NUM_EVENTS];
    if (is_compute_capability_pre_cupti11()) {
        EventName[0] = "cuda:::event:elapsed_cycles_sm:device=0";
    } else {
        EventName[0] = "cuda:::dram__bytes_read.sum:device=0";
    }

    quiet = tests_quiet(argc, argv);

    /* PAPI Initialization */
    retval = PAPI_library_init(PAPI_VER_CURRENT);
    if (retval != PAPI_VER_CURRENT) {
        if (!quiet) {
            printf("PAPI init failed\n");
        }
        test_fail(__FILE__, __LINE__, "PAPI_library_init failed", 0);
    }

    if (!quiet) {
        printf("PAPI_VERSION     : %4d %6d %7d\n",
               PAPI_VERSION_MAJOR(PAPI_VERSION),
               PAPI_VERSION_MINOR(PAPI_VERSION),
               PAPI_VERSION_REVISION(PAPI_VERSION));
    }

#ifdef RUN_WITH_NON_PRIMARY_CONTEXT
    hipError_t cuErr;
    hipCtx_t sessionCtx = NULL;

    cuErr = hipCtxCreate(&sessionCtx, 0, 0);
    if (cuErr != hipSuccess) {
        const char *errString = NULL;
        hipDrvGetErrorString(cuErr, &errString);
        fprintf(stderr, "%s:%s:%i hipCtxCreate hipError_t='%s'.\n",
                __FILE__, __func__, __LINE__, errString);
        test_fail(__FILE__, __LINE__, "hipCtxCreate failed", 0);
    }
#endif

    for (i = 0; i < NUM_EVENTS; i++) {
        retval = PAPI_event_name_to_code((char *)EventName[i], &events[i]);
        if (retval != PAPI_OK) {
            fprintf(stderr, "PAPI_event_name_to_code failed\n");
            continue;
        }
        eventCount++;
        if (!quiet) {
            printf("Name %s --- Code: %#x\n", EventName[i], events[i]);
        }
    }

    if (eventCount == 0) {
        if (!quiet) {
            printf("Test FAILED: no valid events found.\n");
        }
        test_skip(__FILE__, __LINE__, "No events found", 0);
        return 1;
    }

    retval = PAPI_create_eventset(&EventSet);
    if (retval != PAPI_OK) {
        if (!quiet) {
            printf("PAPI_create_eventset failed\n");
        }
        test_fail(__FILE__, __LINE__, "Cannot create eventset", retval);
    }

    /* Use the primary cuda context for the device */
    retval = PAPI_add_events(EventSet, events, eventCount);
    if (retval != PAPI_OK) {
        fprintf(stderr, "PAPI_add_events failed\n");
    }

    retval = PAPI_start(EventSet);
    if (retval != PAPI_OK) {
        fprintf(stderr, "PAPI_start failed\n");
    }

    char str[] = "Hello Cupti!";

    int j;
    for (j = 0; j < 12; j++) {
        /* mangle contents of output */
        str[j] -= j;
    }

    char *d_str;
    size_t size = sizeof(str);
    hipMalloc((void**)&d_str, size);
    hipMemcpy(d_str, str, size, hipMemcpyHostToDevice);

    dim3 dimGrid(2);  // one block per word
    dim3 dimBlock(6); // one thread per character

    helloCupti<<<dimGrid, dimBlock>>>(d_str);

    hipMemcpy(str, d_str, size, hipMemcpyDeviceToHost);
    hipFree(d_str);

    if (!quiet) {
        printf("END: %s\n", str);
    }

    retval = PAPI_stop(EventSet, values);
    if (retval != PAPI_OK) {
        fprintf( stderr, "PAPI_stop failed\n" );
    }

    retval = PAPI_cleanup_eventset(EventSet);
    if (retval != PAPI_OK) {
        fprintf(stderr, "PAPI_cleanup_eventset failed\n");
    }

    retval = PAPI_destroy_eventset(&EventSet);
    if (retval != PAPI_OK) {
        fprintf(stderr, "PAPI_destroy_eventset failed\n");
    }

#ifdef RUN_WITH_NON_PRIMARY_CONTEXT
    hipCtx_t getCtx;
    cuErr = hipCtxPopCurrent(&getCtx);
    if (cuErr != hipSuccess) {
        const char *errString = NULL;
        hipDrvGetErrorString(cuErr, &errString);
        fprintf(stderr, "%s:%s:%i hipCtxPopCurrent hipError_t='%s'.\n",
                __FILE__, __func__, __LINE__, errString);
        test_fail(__FILE__,__LINE__, "hipCtxPopCurrent failed", 0);
    }
    cuErr = hipCtxDestroy(sessionCtx);
    if (cuErr != hipSuccess) {
        const char *errString = NULL;
        hipDrvGetErrorString(cuErr, &errString);
        fprintf(stderr, "%s:%s:%i hipCtxDestroy hipError_t='%s'.\n",
                __FILE__, __func__, __LINE__, errString);
        test_fail(__FILE__,__LINE__, "hipCtxDestroy failed", 0);
    }
#endif

    PAPI_shutdown();

    for (i = 0; i < eventCount; i++) {
        if (!quiet) {
            printf("%12lld \t\t --> %s \n", values[i], EventName[i]);
        }
    }

    test_pass(__FILE__);

    return 0;
}

__global__ void
helloCupti(char* str)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    str[idx] += idx;
}
