#include "hip/hip_runtime.h"
/****************************/
/* THIS IS OPEN SOURCE CODE */
/****************************/

/**
 * @file    hello_cupti.cu
 * @author  Heike Jagode
 *          jagode@eecs.utk.edu
 * Mods:    Giuseppe Congiu
 *          gcongiu@icl.utk.edu
 * test case for Example component
 *
 *
 * @brief
 *  This file is a very simple HelloWorld C example which serves (together
 *  with its Makefile) as a guideline on how to add tests to components.
 *  The papi configure and papi Makefile will take care of the compilation
 *  of the component tests (if all tests are added to a directory named
 *  'tests' in the specific component dir).
 *  See components/README for more details.
 *
 *  The string "Hello Cupti!" is mangled and then restored.
 */

#include <stdio.h>
#include "papi.h"
#include "papi_test.h"
#include "common.h"

#define NUM_EVENTS 1

// Prototypes
__global__ void helloCupti(char*);

// Host function
int main(int argc, char** argv)
{
    int retval, i;
    int EventSet = PAPI_NULL;
    long long values[NUM_EVENTS];
    int events[NUM_EVENTS];
    int eventCount = 0;
    int quiet;

    quiet = tests_quiet(argc, argv);

    /* PAPI Initialization */
    retval = PAPI_library_init(PAPI_VER_CURRENT);
    if (retval != PAPI_VER_CURRENT) {
        if (!quiet) {
            printf("PAPI init failed\n");
        }
        test_fail(__FILE__, __LINE__, "PAPI_library_init failed", 0);
    }

    if (!quiet) {
        printf("PAPI_VERSION     : %4d %6d %7d\n",
               PAPI_VERSION_MAJOR(PAPI_VERSION),
               PAPI_VERSION_MINOR(PAPI_VERSION),
               PAPI_VERSION_REVISION(PAPI_VERSION));
    }

    int cid = get_cuda_component_id();
    events[0] = 0 | PAPI_NATIVE_MASK;
    retval = PAPI_enum_cmp_event(&events[0], PAPI_ENUM_FIRST, cid);
    if (retval != PAPI_OK) {
        test_fail(__FILE__, __LINE__, "PAPI_enum_cmp_event failed.", retval);
    }

    PAPI_event_info_t event_info;
    retval = PAPI_get_event_info(events[0], &event_info);
    if (retval != PAPI_OK) {
        test_fail(__FILE__, __LINE__, "PAPI_get_event_info failed.", retval);
    }

    const char *EventName[NUM_EVENTS];
    EventName[0] = event_info.symbol;

#ifdef RUN_WITH_NON_PRIMARY_CONTEXT
    hipError_t cuErr;
    hipCtx_t sessionCtx = NULL;

    cuErr = hipCtxCreate(&sessionCtx, 0, 0);
    if (cuErr != hipSuccess) {
        const char *errString = NULL;
        hipDrvGetErrorString(cuErr, &errString);
        fprintf(stderr, "%s:%s:%i hipCtxCreate hipError_t='%s'.\n",
                __FILE__, __func__, __LINE__, errString);
        test_fail(__FILE__, __LINE__, "hipCtxCreate failed", 0);
    }
#endif

    for (i = 0; i < NUM_EVENTS; i++) {
        eventCount++;
        if (!quiet) {
            printf("Name %s --- Code: %#x\n", EventName[i], events[i]);
        }
    }

    if (eventCount == 0) {
        if (!quiet) {
            printf("Test FAILED: no valid events found.\n");
        }
        test_skip(__FILE__, __LINE__, "No events found", 0);
        return 1;
    }

    retval = PAPI_create_eventset(&EventSet);
    if (retval != PAPI_OK) {
        if (!quiet) {
            printf("PAPI_create_eventset failed\n");
        }
        test_fail(__FILE__, __LINE__, "Cannot create eventset", retval);
    }

    /* Use the primary cuda context for the device */
    retval = PAPI_add_events(EventSet, events, eventCount);
    if (retval != PAPI_OK) {
        fprintf(stderr, "PAPI_add_events failed\n");
    }

    retval = PAPI_start(EventSet);
    if (retval != PAPI_OK) {
        fprintf(stderr, "PAPI_start failed\n");
    }

    char str[] = "Hello Cupti!";

    int j;
    for (j = 0; j < 12; j++) {
        /* mangle contents of output */
        str[j] -= j;
    }

    char *d_str;
    size_t size = sizeof(str);
    hipMalloc((void**)&d_str, size);
    hipMemcpy(d_str, str, size, hipMemcpyHostToDevice);

    dim3 dimGrid(2);  // one block per word
    dim3 dimBlock(6); // one thread per character

    helloCupti<<<dimGrid, dimBlock>>>(d_str);

    hipMemcpy(str, d_str, size, hipMemcpyDeviceToHost);
    hipFree(d_str);

    if (!quiet) {
        printf("END: %s\n", str);
    }

    retval = PAPI_stop(EventSet, values);
    if (retval != PAPI_OK) {
        fprintf( stderr, "PAPI_stop failed\n" );
    }

    retval = PAPI_cleanup_eventset(EventSet);
    if (retval != PAPI_OK) {
        fprintf(stderr, "PAPI_cleanup_eventset failed\n");
    }

    retval = PAPI_destroy_eventset(&EventSet);
    if (retval != PAPI_OK) {
        fprintf(stderr, "PAPI_destroy_eventset failed\n");
    }

#ifdef RUN_WITH_NON_PRIMARY_CONTEXT
    hipCtx_t getCtx;
    cuErr = hipCtxPopCurrent(&getCtx);
    if (cuErr != hipSuccess) {
        const char *errString = NULL;
        hipDrvGetErrorString(cuErr, &errString);
        fprintf(stderr, "%s:%s:%i hipCtxPopCurrent hipError_t='%s'.\n",
                __FILE__, __func__, __LINE__, errString);
        test_fail(__FILE__,__LINE__, "hipCtxPopCurrent failed", 0);
    }
    cuErr = hipCtxDestroy(sessionCtx);
    if (cuErr != hipSuccess) {
        const char *errString = NULL;
        hipDrvGetErrorString(cuErr, &errString);
        fprintf(stderr, "%s:%s:%i hipCtxDestroy hipError_t='%s'.\n",
                __FILE__, __func__, __LINE__, errString);
        test_fail(__FILE__,__LINE__, "hipCtxDestroy failed", 0);
    }
#endif

    PAPI_shutdown();

    for (i = 0; i < eventCount; i++) {
        if (!quiet) {
            printf("%12lld \t\t --> %s \n", values[i], EventName[i]);
        }
    }

    test_pass(__FILE__);

    return 0;
}

__global__ void
helloCupti(char* str)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    str[idx] += idx;
}
