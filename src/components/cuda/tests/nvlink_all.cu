#include "hip/hip_runtime.h"
/*
 * Copyright 2015-2016 NVIDIA Corporation. All rights reserved.
 *
 * Sample to demonstrate use of NVlink CUPTI APIs
 *
 * This version is significantly changed to use PAPI and the CUDA component to
 * handle access and reporting. As of 10/05/2018, I have deleted all CUPTI_ONLY
 * references, for clarity. The file nvlink_bandwidth_cupti_only.cu contains
 * the cupti-only code.  I also deleted the #if PAPI; there is no option
 * without PAPI.  Also, before my changes, the makefile did not even have a
 * build option that set CUPTI_ONLY for this file.
 *
 * -TonyC.
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include "papi.h"
#include "papi_test.h"

#define MAX_DEVICES    (32)
#define BLOCK_SIZE     (1024)
#define GRID_SIZE      (512)
#define BUF_SIZE       (32 * 1024)
#define ALIGN_SIZE     (8)
#define SUCCESS        (0)
#define NUM_METRIC     (18)
#define NUM_EVENTS     (2)
#define MAX_SIZE       (64*1024*1024)   // 64 MB

typedef union
{
    long long ll;
    unsigned long long ull;
    double    d;
    void *vp;
    unsigned char ch[8];
} convert_64_t;

typedef struct {
    char name[128];
    long long value;
} eventStore_t;

int eventsFoundCount = 0;               // occupants of the array.
int eventsFoundMax;                     // Size of the array.
int eventsFoundAdd = 32;                // Blocksize for increasing the array.
eventStore_t *eventsFound = NULL;       // The array.

int Streams;                            // Gets asyncEngineCount (number of physical copy engines).
int cpuToGpu = 0;
int gpuToGpu = 0;
size_t bufferSize = 0;

int         *deviceEvents = NULL;
hipDeviceptr_t *pDevBuffer0  = NULL;
hipDeviceptr_t *pDevBuffer1  = NULL;
float       **pHostBuffer = NULL;
hipStream_t *cudaStreams = NULL;

extern "C" __global__ void test_nvlink_bandwidth(float *source, float *dest)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    dest[idx] = source[idx] * 2.0f;
}

#define DIM(x) (sizeof(x)/sizeof(*(x))) /* compute elements in an array */

void FreeGlobals(void)
{
    int i;
    hipError_t cuda_errno = hipSuccess;
    free(deviceEvents);

    for (i = 0; i < Streams; i++) {
        cuda_errno = hipSetDevice(0);
        if (cuda_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipSetDevice failed.", 0);
        }

        cuda_errno = hipFree((void **) &pDevBuffer0[i]);
        if (cuda_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipFree failed.", 0);
        }

        free(pHostBuffer[i]);
    }

    free(pDevBuffer0);
    free(pHostBuffer);
    free(pDevBuffer1);

    for (i = 0; i < Streams; i++) {
        if (cudaStreams[i] != NULL) {
            cuda_errno = hipStreamDestroy(cudaStreams[i]);
            if (cuda_errno != hipSuccess) {
                test_fail(__FILE__, __LINE__, "hipStreamDestroy failed.", 0);
            }
        }
    }

    free(cudaStreams);
}

void calculateSize(char *result, uint64_t size)
{
    int i;

    const char *sizes[] = { "TB", "GB", "MB", "KB", "B" };
    uint64_t exbibytes = 1024ULL * 1024ULL * 1024ULL * 1024ULL;

    uint64_t multiplier = exbibytes;

    for (i = 0; (unsigned) i < DIM(sizes); i++, multiplier /= (uint64_t) 1024) {
        if (size < multiplier)
            continue;
        sprintf(result, "%.1f %s", (float) size / multiplier, sizes[i]);
        return;
    }
    strcpy(result, "0");
    return;
}

void testCpuToGpu(hipDeviceptr_t * pDevBuffer, float **pHostBuffer, size_t bufferSize,
                  hipStream_t * cudaStreams)
{
    int i;
    hipError_t cuda_errno = hipSuccess;

    for (i = 0; i < Streams; i++) {
        cuda_errno = hipMemcpyAsync((void *) pDevBuffer[i], pHostBuffer[i],
                                     bufferSize, hipMemcpyHostToDevice,
                                     cudaStreams[i]);
        if (cuda_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipMemcpyHostToDevice failed", 0);
        }
    }

    cuda_errno = hipDeviceSynchronize();
    if (cuda_errno != hipSuccess) {
        test_fail(__FILE__, __LINE__, "hipDeviceSynchronize failed.", 0);
    }

    for (i = 0; i < Streams; i++) {
        cuda_errno = hipMemcpyAsync(pHostBuffer[i], (void *) pDevBuffer[i],
                                     bufferSize, hipMemcpyDeviceToHost,
                                     cudaStreams[i]);
    }

    cuda_errno = hipDeviceSynchronize();
    if (cuda_errno != hipSuccess) {
        test_fail(__FILE__, __LINE__, "hipMemcpyAsync failed.", 0);
    }

    for (i = 0; i < Streams; i += 2) {
        cuda_errno = hipMemcpyAsync((void *) pDevBuffer[i], pHostBuffer[i],
                                     bufferSize, hipMemcpyHostToDevice,
                                     cudaStreams[i]);
        if (cuda_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipMemcpyAsync failed.", 0);
        }

        cuda_errno = hipMemcpyAsync(pHostBuffer[i + 1], (void *) pDevBuffer[i + 1],
                                     bufferSize, hipMemcpyDeviceToHost,
                                     cudaStreams[i + 1]);
        if (cuda_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipMemcpyAsync failed.", 0);
        }
    }

    cuda_errno = hipDeviceSynchronize();
    if (cuda_errno != hipSuccess) {
        test_fail(__FILE__, __LINE__, "hipDeviceSynchronize failed.", 0);
    }
}

void testGpuToGpu_part1(hipDeviceptr_t * pDevBuffer0, hipDeviceptr_t * pDevBuffer1,
                        float **pHostBuffer, size_t bufferSize,
                        hipStream_t * cudaStreams)
{
    int i;
    hipError_t cuda_errno = hipSuccess;

    for (i = 0; i < Streams; i++) {
        cuda_errno = hipMemcpyAsync((void *) pDevBuffer0[i], pHostBuffer[i],
                                     bufferSize, hipMemcpyHostToDevice,
                                     cudaStreams[i]);
        if (cuda_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipMemcpyAsync failed.", 0);
        }
    }

    cuda_errno = hipDeviceSynchronize();
    if (cuda_errno != hipSuccess) {
        test_fail(__FILE__, __LINE__, "hipDeviceSynchronize failed.", 0);
    }

    for (i = 0; i < Streams; i++) {
        cuda_errno = hipMemcpyAsync((void *) pDevBuffer1[i], pHostBuffer[i],
                                     bufferSize, hipMemcpyHostToDevice,
                                     cudaStreams[i]);
        if (cuda_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "cudaMemcpuAsync failed.", 0);
        }
    }

    cuda_errno = hipDeviceSynchronize();
    if (cuda_errno != hipSuccess) {
        test_fail(__FILE__, __LINE__, "hipDeviceSynchronize failed.", 0);
    }
}

void testGpuToGpu_part2(hipDeviceptr_t * pDevBuffer0, hipDeviceptr_t * pDevBuffer1,
                        float **pHostBuffer, size_t bufferSize,
                        hipStream_t * cudaStreams)
{
    int i;
    hipError_t cuda_errno = hipSuccess;

    for (i = 0; i < Streams; i++) {
        cuda_errno = hipMemcpyAsync((void *) pDevBuffer0[i], (void *) pDevBuffer1[i],
                                     bufferSize, hipMemcpyDeviceToDevice,
                                     cudaStreams[i]);
        if (cuda_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipMemcpyAsync failed.", 0);
        }
    }

    cuda_errno = hipDeviceSynchronize();
    if (cuda_errno != hipSuccess) {
        test_fail(__FILE__, __LINE__, "hipDeviceSynchronize failed.", 0);
    }

    for(i = 0; i < Streams; i++) {
        cuda_errno = hipMemcpyAsync((void *) pDevBuffer1[i], (void *) pDevBuffer0[i],
                                     bufferSize, hipMemcpyDeviceToDevice,
                                     cudaStreams[i]);
        if (cuda_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipMemcpyAsync failed.", 0);
        }
    }

    cuda_errno = hipDeviceSynchronize();
    if (cuda_errno != hipSuccess) {
        test_fail(__FILE__, __LINE__, "hipDeviceSynchronize failed.", 0);
    }

    for(i = 0; i < Streams; i++) {
        test_nvlink_bandwidth<<<GRID_SIZE, BLOCK_SIZE>>>((float *) pDevBuffer1[i],
                                                         (float *) pDevBuffer0[i]);
    }
}

void conductCpuToGpu(int EventSet, int device, long long *values)
{
    int i;
    int papi_errno = PAPI_OK;
    hipError_t cuda_errno = hipSuccess;

    if (device == 0) {
        papi_errno = PAPI_start(EventSet);
        if (papi_errno != PAPI_OK) {
            test_fail(__FILE__, __LINE__, "PAPI_start failed.", papi_errno);
        }

        testCpuToGpu(pDevBuffer0, pHostBuffer, bufferSize, cudaStreams);
    } else {
        cuda_errno = hipSetDevice(device);
        if (cuda_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipSetDevice failed.", 0);
        }

        for(i = 0; i < Streams; i++) {
            cuda_errno = hipMalloc((void **) &pDevBuffer1[i], bufferSize);
            if (cuda_errno != hipSuccess) {
                test_fail(__FILE__, __LINE__, "hipMalloc failed.", 0);
            }
        }

        papi_errno = PAPI_start(EventSet);
        if (papi_errno != PAPI_OK) {
            test_fail(__FILE__, __LINE__, "PAPI_start failed.", papi_errno);
        }

        testCpuToGpu(pDevBuffer1, pHostBuffer, bufferSize, cudaStreams);

        for (i = 0; i < Streams; i++) {
            cuda_errno = hipFree((void **) pDevBuffer1[i]);
            if (cuda_errno != hipSuccess) {
                test_fail(__FILE__, __LINE__, "hipFree failed.", 0);
            }
        }
    }

    papi_errno = PAPI_stop(EventSet, values);
    if (papi_errno != PAPI_OK) {
        test_fail(__FILE__, __LINE__, "PAPI_stop failed.", papi_errno);
    }
}

void conductGpuToGpu(int EventSet, int device, long long *values) {
    int i;
    int partner = device;
    hipError_t cuda_errno = hipSuccess;
    int papi_errno = PAPI_OK;

    if (device == 0) {
        partner = 1;
    }

    cuda_errno = hipSetDevice(0);
    if (cuda_errno != hipSuccess) {
        test_fail(__FILE__, __LINE__, "hipSetDevice failed.", 0);
    }

    cuda_errno = hipDeviceEnablePeerAccess(partner, 0);
    if (cuda_errno != hipSuccess) {
        test_fail(__FILE__, __LINE__, "hipDeviceEnablePeerAccess failed.", 0);
    }

    cuda_errno = hipSetDevice(partner);
    if (cuda_errno != hipSuccess) {
        test_fail(__FILE__, __LINE__, "hipSetDevice failed.", 0);
    }

    cuda_errno = hipDeviceEnablePeerAccess(0, 0);
    if (cuda_errno != hipSuccess) {
        test_fail(__FILE__, __LINE__, "hipDeviceEnablePeerAccess failed.", 0);
    }

    for (i = 0; i < Streams; i++) {
        cuda_errno = hipMalloc((void **) &pDevBuffer1[i], bufferSize);
        if (cuda_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipMalloc failed.", 0);
        }
    }

    testGpuToGpu_part1(pDevBuffer0, pDevBuffer1, pHostBuffer, bufferSize,
                       cudaStreams);

    papi_errno = PAPI_start(EventSet);
    if (papi_errno != PAPI_OK) {
        test_fail(__FILE__, __LINE__, "PAPI_start failed.", papi_errno);
    }

    testGpuToGpu_part2(pDevBuffer0, pDevBuffer1, pHostBuffer, bufferSize,
                       cudaStreams);

    papi_errno = PAPI_stop(EventSet, values);
    if (papi_errno != PAPI_OK) {
        test_fail(__FILE__, __LINE__, "PAPI_stop failed.", papi_errno);
    }

    cuda_errno = hipSetDevice(0);
    if (cuda_errno != hipSuccess) {
        test_fail(__FILE__, __LINE__, "hipSetDevice failed.", 0);
    }

    cuda_errno = hipDeviceDisablePeerAccess(partner);
    if (cuda_errno != hipSuccess) {
        test_fail(__FILE__, __LINE__, "hipDeviceDisablePeerAccess failed.", 0);
    }

    cuda_errno = hipSetDevice(partner);
    if (cuda_errno != hipSuccess) {
        test_fail(__FILE__, __LINE__, "hipSetDevice failed.", 0);
    }

    cuda_errno = hipDeviceDisablePeerAccess(0);
    if (cuda_errno != hipSuccess) {
        test_fail(__FILE__, __LINE__, "hipDeviceDisablePeerAccess failed.", 0);
    }

    for (i = 0; i < Streams; i++) {
        cuda_errno = hipFree((void **) pDevBuffer1[i]);
        if (cuda_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipFree failed.", 0);
        }
    }

    cuda_errno = hipSetDevice(0);
    if (cuda_errno != hipSuccess) {
        test_fail(__FILE__, __LINE__, "hipSetDevice failed.", 0);
    }
}

static void printUsage()
{
    printf("usage: Demonstrate use of NVlink CUPTI APIs\n");
    printf("    -h, -help, --help: display this help message.\n");
    printf("    Otherwise, exactly one of these options:\n");
    printf("    --cpu-to-gpu: Show results for data transfer between CPU and GPU.\n");
    printf("    --gpu-to-gpu: Show results for data transfer between two GPUs.\n");
}

void parseCommandLineArgs(int argc, char *argv[])
{
    if (argc != 2) {
        printf("Invalid number of options\n");
        printUsage();
        exit(EXIT_SUCCESS);
    }

    if (strcmp(argv[1], "--cpu-to-gpu") == 0) {
        cpuToGpu = 1;
    } else if (strcmp(argv[1], "--gpu-to-gpu") == 0) {
        gpuToGpu = 1;
    } else if ((strcmp(argv[1], "--help") == 0) ||
               (strcmp(argv[1], "-help") == 0)  ||
               (strcmp(argv[1], "-h") == 0)) {
        printUsage();
        exit(EXIT_SUCCESS);
    } else {
        printf("Failed to understand argument '%s'.\n", argv[1]);
        printUsage();
        exit(EXIT_FAILURE);
    }
}

void addEventsFound(char *eventName, long long value)
{
    strncpy(eventsFound[eventsFoundCount].name, eventName, 127);
    eventsFound[eventsFoundCount].value = value;

    if (++eventsFoundCount >= eventsFoundMax) {
        eventsFoundMax += eventsFoundAdd;
        eventsFound =
            (eventStore_t*) realloc(eventsFound,
                                    eventsFoundMax * sizeof(eventStore_t));
        memset(eventsFound+(eventsFoundMax - eventsFoundAdd), 0,
               eventsFoundAdd * sizeof(eventStore_t));
    }
}

int main(int argc, char *argv[])
{
    int device, deviceCount = 0, i = 0;
    size_t freeMemory = 0, totalMemory = 0;
    char str[64];
    int papi_errno, cu_errno;
    hipError_t cuda_errno;

    eventsFoundMax = eventsFoundAdd;
    eventsFound = (eventStore_t*) calloc(eventsFoundMax, sizeof(eventStore_t));

    hipDeviceProp_t prop[MAX_DEVICES];

    // Parse command line arguments
    parseCommandLineArgs(argc, argv);

    cu_errno = hipInit(0);
    if (cu_errno != hipSuccess) {
        test_fail(__FILE__, __LINE__, "hipInit failed.", 0);
    }

    cuda_errno = hipGetDeviceCount(&deviceCount);
    if (cuda_errno != hipSuccess) {
        test_fail(__FILE__, __LINE__, "hipGetDeviceCount failed.", 0);
    }

    printf("There are %d devices.\n", deviceCount);

    if(deviceCount == 0) {
        printf("There is no device supporting CUDA.\n");
        exit(EXIT_FAILURE);
    }

    Streams = 1;
    for (device = 0; device < deviceCount; device++) {
        cuda_errno = hipGetDeviceProperties(&prop[device], device);
        if (cuda_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipGetDeviceProperties failed.", 0);
        }

        printf("CUDA Device %d Name: %s", i, prop[i].name);
        printf(", AsyncEngineCount=%i", prop[i].asyncEngineCount);
        printf(", MultiProcessors=%i", prop[i].multiProcessorCount);
        printf(", MaxThreadsPerMP=%i", prop[i].maxThreadsPerMultiProcessor);
        printf("\n");

        if (prop[i].asyncEngineCount > Streams) {
            Streams = prop[i].asyncEngineCount;
        }
    }

    printf("Streams to use: %i (= max Copy Engines).\n", Streams);

    deviceEvents= (int*)          calloc(deviceCount, sizeof(int));
    pDevBuffer0 = (hipDeviceptr_t*)  calloc(Streams, sizeof(hipDeviceptr_t));
    pDevBuffer1 = (hipDeviceptr_t*)  calloc(Streams, sizeof(hipDeviceptr_t));
    pHostBuffer = (float **)      calloc(Streams, sizeof(float*));
    cudaStreams = (hipStream_t*) calloc(Streams, sizeof(hipStream_t));

    cuda_errno = hipMemGetInfo(&freeMemory, &totalMemory);
    if (cuda_errno != hipSuccess) {
        test_fail(__FILE__, __LINE__, "hipMemGetInfo failed.", 0);
    }

    printf("Total Device Memory available : ");
    calculateSize(str, (uint64_t) totalMemory);
    printf("%s\n", str);

    bufferSize = MAX_SIZE < (freeMemory / 4) ? MAX_SIZE : (freeMemory / 4);
    bufferSize = bufferSize/2;
    printf("Memcpy size is set to %llu B (%llu MB)\n",
           (unsigned long long) bufferSize,
           (unsigned long long) bufferSize / (1024 * 1024));

    for (i = 0; i < Streams; i++) {
        cuda_errno = hipStreamCreate(&cudaStreams[i]);
        if (cuda_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipStreamCreate failed.", 0);
        }
    }

    cuda_errno = hipDeviceSynchronize();
    if (cuda_errno != hipSuccess) {
        test_fail(__FILE__, __LINE__, "hipDeviceSynchronize failed.", 0);
    }

    int EventSet = PAPI_NULL;
    int eventCount;
    int retval;
    int k, m, cid = -1;

    /* PAPI Initialization */
    retval = PAPI_library_init(PAPI_VER_CURRENT);
    if(retval != PAPI_VER_CURRENT) {
        fprintf(stderr, "PAPI_library_init failed, ret=%i [%s]\n",
                retval, PAPI_strerror(retval));
        FreeGlobals();
        exit(EXIT_FAILURE);
    }

    printf("PAPI version: %d.%d.%d\n", PAPI_VERSION_MAJOR(PAPI_VERSION),
                                       PAPI_VERSION_MINOR(PAPI_VERSION),
                                       PAPI_VERSION_REVISION(PAPI_VERSION));

    k = PAPI_num_components();
    for (i = 0; i < k && cid < 0; i++) {
        PAPI_component_info_t *aComponent = 
            (PAPI_component_info_t*) PAPI_get_component_info(i);
        if (aComponent == NULL) {
            fprintf(stderr, "PAPI_get_component_info(%i) failed, "
                    "returned NULL. %i components reported.\n", i, k);
            FreeGlobals();
            exit(EXIT_FAILURE);
        }

       if (strcmp("cuda", aComponent->name) == 0) cid = i;
    }

    if (cid < 0) {
        fprintf(stderr, "Failed to find cuda component among %i "
               "reported components.\n", k);
        FreeGlobals();
        PAPI_shutdown();
        exit(EXIT_FAILURE);
    }

    printf("Found CUDA Component at id %d\n", cid);

    eventCount = 0;
    int eventsRead = 0;

    for(i = 0; i < Streams; i++) {
        cuda_errno = hipMalloc((void **) &pDevBuffer0[i], bufferSize);
        if (cuda_errno != hipSuccess) {
            test_fail(__FILE__, __LINE__, "hipMalloc failer.", 0);
        }

        pHostBuffer[i] = (float *) malloc(bufferSize);
        if (pHostBuffer[i] == NULL) {
            test_fail(__FILE__, __LINE__, "malloc failed.", 0);
        }
    }

    if (cpuToGpu) {
       printf("Experiment timing memory copy from host to GPU.\n");
    }

    if (gpuToGpu) {
        printf("Experiment timing memory copy between GPU 0 and each other GPU.\n");
    }

   printf("Events with numeric values were read; if they are zero, they may not  \n"
          "be operational, or the exercises performed by this code do not affect \n"
          "them. We report all 'nvlink' events presented by the cuda component.  \n"
          "\n"
          "---------------------------Event Name---------------------------:---Value---\n");

    PAPI_event_info_t info;
    m = PAPI_NATIVE_MASK;
    papi_errno = PAPI_enum_cmp_event(&m, PAPI_ENUM_FIRST, cid);
    do {
        memset(&info,0,sizeof(PAPI_event_info_t));
        k = m;

        do {
            papi_errno = PAPI_get_event_info(k, &info);

            if (strstr(info.symbol, "nvlink") == NULL)
                continue;
            char *devstr = strstr(info.symbol, "device=");
            if (devstr == NULL)
                continue;
            device = atoi(devstr + 7);
            if (device < 0 || device >= deviceCount)
                continue;
            deviceEvents[device]++;

            papi_errno = PAPI_create_eventset(&EventSet);
            if (papi_errno != PAPI_OK) {
                test_fail(__FILE__, __LINE__, "PAPI_create_eventset failed.",
                          papi_errno);
            }

            papi_errno = PAPI_assign_eventset_component(EventSet, cid);
            if (papi_errno != PAPI_OK) {
                test_fail(__FILE__, __LINE__, "PAPI_assign_eventset_component failed.",
                          papi_errno);
            }

            papi_errno = PAPI_add_named_event(EventSet, info.symbol);
            if(papi_errno == PAPI_OK) {
                eventCount++;
            } else {
                papi_errno = PAPI_cleanup_eventset(EventSet);
                if (papi_errno != PAPI_OK) {
                    test_fail(__FILE__, __LINE__, "PAPI_cleanup_eventset failed.",
                              papi_errno);
                }

                papi_errno = PAPI_destroy_eventset(&EventSet);
                if (papi_errno != PAPI_OK) {
                    test_fail(__FILE__, __LINE__, "PAPI_destroy_eventset failed.",
                              papi_errno);
                }
                continue;
            }

            long long value = -1;

            if(cpuToGpu) {
                conductCpuToGpu(EventSet, device, &value);
            } else if(gpuToGpu) {
                conductGpuToGpu(EventSet, device, &value);
            }

            addEventsFound(info.symbol, value);

            papi_errno = PAPI_cleanup_eventset(EventSet);
            if (papi_errno != PAPI_OK) {
                test_fail(__FILE__, __LINE__, "PAPI_cleanup_eventset failed.", 0);
            }

            papi_errno = PAPI_destroy_eventset(&EventSet);
            if (papi_errno != PAPI_OK) {
                test_fail(__FILE__, __LINE__, "PAPI_destroy_eventset failed.", 0);
            }

            if (value >= 0) {
                eventsRead++;
                calculateSize(str, value);
                if (value == 0) {
                    printf("%-64s: %9s (not exercised by current test code.)\n",
                           info.symbol, str);
                } else {
                    printf("%-64s: %9s\n", info.symbol, str);
                }
            } else {
                printf("%-64s: Failed to read.\n", info.symbol);
            }
        } while (PAPI_enum_cmp_event(&k, PAPI_NTV_ENUM_UMASKS, cid) == PAPI_OK);
    } while (PAPI_enum_cmp_event(&m, PAPI_ENUM_EVENTS, cid) == PAPI_OK);

    if (eventCount < 1) {
        fprintf(stderr, "Unable to add any NVLINK events; they are not present in the component.\n");
        fprintf(stderr, "Unable to proceed with this test.\n");
        FreeGlobals();
        PAPI_shutdown();
        exit(EXIT_FAILURE);
    }

    if (eventsRead < 1) {
        printf("\nFailed to read any nvlink events.\n");
        fprintf(stderr, "Unable to proceed with this test.\n");
        FreeGlobals();
        PAPI_shutdown();
        exit(EXIT_FAILURE);
    }

    printf("\nTotal nvlink events identified: %i.\n\n", eventsFoundCount);
    if (eventsFoundCount < 2) {
        printf("Insufficient events are exercised by the current test code to perform pair testing.\n");
        FreeGlobals();
        PAPI_shutdown();
        exit(EXIT_SUCCESS);
    }

    for (i = 0; i < deviceCount; i++) {
        printf("Device %i has %i events. %i potential pairings per device.\n",
               i, deviceEvents[i], deviceEvents[i] * (deviceEvents[i]-1) / 2);
    }

    // Begin pair testing. We consider every possible pairing of events
    // that, tested alone, returned a value greater than zero.

    int mainEvent, pairEvent, mainDevice, pairDevice;
    long long saveValues[2];
    long long readValues[2];
    int  goodOnSame = 0, failOnDiff = 0, badSameCombo = 0, pairProblems = 0;
    int type;
    for (type = 0; type < 2; type++) {
        if (type == 0) {
            printf("List of Pairings on SAME device:\n");
            printf("* means value changed by more than 10%% when paired (vs measured singly, above).\n");
            printf("^ means a pair was rejected as an invalid combo.\n");
        } else {
            printf("List of Failed Pairings on DIFFERENT devices:\n");
        }

        for (mainEvent = 0; mainEvent < eventsFoundCount - 1; mainEvent++) {
             char *devstr = strstr(eventsFound[mainEvent].name, "device=");
             mainDevice = atoi(devstr + 7);

            for (pairEvent = mainEvent + 1; pairEvent < eventsFoundCount; pairEvent++) {
                devstr = strstr(eventsFound[pairEvent].name, "device=");
                pairDevice=atoi(devstr + 7);

                if (type == 0 && mainDevice != pairDevice)
                    continue;
                if (type == 1 && mainDevice == pairDevice)
                    continue;

                papi_errno = PAPI_create_eventset(&EventSet);
                if (papi_errno != PAPI_OK) {
                    test_fail(__FILE__, __LINE__, "PAPI_create_eventset failed.",
                              papi_errno);
                }

                papi_errno = PAPI_assign_eventset_component(EventSet, cid);
                if (papi_errno != PAPI_OK) {
                    test_fail(__FILE__, __LINE__, "PAPI_assign_eventset_component failed.",
                              papi_errno);
                }

                papi_errno = PAPI_add_named_event(EventSet, eventsFound[mainEvent].name);
                if (papi_errno != PAPI_OK) {
                    test_fail(__FILE__, __LINE__, "PAPI_add_named_event failed.",
                              papi_errno);
                }

                papi_errno = PAPI_add_named_event(EventSet, eventsFound[pairEvent].name);
                if (type == 0 && papi_errno == PAPI_ECOMBO) {
                    printf("%c %64s + %-64s [Invalid Combo]\n", '^',
                        eventsFound[mainEvent].name, eventsFound[pairEvent].name);
                    badSameCombo++;
                    papi_errno = PAPI_cleanup_eventset(EventSet);
                    if (papi_errno != PAPI_OK) {
                        test_fail(__FILE__, __LINE__, "PAPI_cleanup_eventset failed.",
                                  papi_errno);
                    }

                    papi_errno = PAPI_destroy_eventset(&EventSet);
                    if (papi_errno != PAPI_OK) {
                        test_fail(__FILE__, __LINE__, "PAPI_destroy_eventset failed.",
                                  papi_errno);
                    }
                    continue;
                }

                if (type == 1 && papi_errno == PAPI_ECOMBO) {
                    printf("%64s + %-64s BAD COMBINATION ACROSS DEVICES.\n",
                        eventsFound[mainEvent].name, eventsFound[pairEvent].name);
                    failOnDiff++;
                    papi_errno = PAPI_cleanup_eventset(EventSet);
                    if (papi_errno != PAPI_OK) {
                        test_fail(__FILE__, __LINE__, "PAPI_cleanup_eventset failed.",
                                  papi_errno);
                    }

                    papi_errno = PAPI_destroy_eventset(&EventSet);
                    if (papi_errno != PAPI_OK) {
                        test_fail(__FILE__, __LINE__, "PAPI_destroy_eventset failed.",
                                  papi_errno);
                    }
                    continue;
                }

                if (papi_errno != PAPI_OK) {
                    fprintf(stderr, "%s:%d Attempt to add event '%s' to set "
                            "with event '%s' produced an unexpected error: "
                            "[%s]. Ignoring this pair.\n",
                            __FILE__, __LINE__, eventsFound[pairEvent],
                            eventsFound[mainEvent], PAPI_strerror(papi_errno));
                    papi_errno = PAPI_cleanup_eventset(EventSet);
                    if (papi_errno != PAPI_OK) {
                        test_fail(__FILE__, __LINE__, "PAPI_cleanup_eventset failed.",
                                  papi_errno);
                    }

                    papi_errno = PAPI_destroy_eventset(&EventSet);
                    if (papi_errno != PAPI_OK) {
                        test_fail(__FILE__, __LINE__, "PAPI_destroy_eventset failed.",
                                  papi_errno);
                    }
                    continue;
                }

                // We were able to add the pair. In type 1, we just skip it,
                // because we presume a single event on a device isn't changed
                // by any event on another device.
                if (type == 1) {
                    papi_errno = PAPI_cleanup_eventset(EventSet);
                    if (papi_errno != PAPI_OK) {
                        test_fail(__FILE__, __LINE__, "PAPI_cleanup_eventset failed.",
                                  papi_errno);
                    }

                    papi_errno = PAPI_destroy_eventset(&EventSet);
                    if (papi_errno != PAPI_OK) {
                        test_fail(__FILE__, __LINE__, "PAPI_destroy_eventset failed.",
                                  papi_errno);
                    }
                    continue;
                }

                // We were able to add the pair, in type 0, get a measurement.
                readValues[0] = -1; readValues[1] = -1;

                if(cpuToGpu) {
                    conductCpuToGpu(EventSet, mainDevice, readValues);
                    saveValues[0] = readValues[0];
                    saveValues[1] = readValues[1];
                } else if(gpuToGpu) {
                    conductGpuToGpu(EventSet, mainDevice, readValues);
                    saveValues[0] = readValues[0];
                    saveValues[1] = readValues[1];
                }

                goodOnSame++;

                // For the checks, we add 2 (so -1 becomes +1) to avoid any
                // divide by zeros. It won't make a significant difference
                // in the ratios. (none if readings are the same).
                double mainSingle = (2.0 + eventsFound[mainEvent].value);
                double pairSingle = (2.0 + eventsFound[pairEvent].value);
                double mainCheck  = mainSingle/(2.0 + saveValues[0]);
                double pairCheck  = pairSingle/(2.0 + saveValues[1]);

                char flag=' ', flag1=' ', flag2=' ';
                if (mainCheck < 0.90 || mainCheck > 1.10) flag1='*';
                if (pairCheck < 0.90 || pairCheck > 1.10) flag2='*';
                if (flag1 == '*' || flag2 == '*') {
                    pairProblems++;
                    flag = '*';
                }

                printf("%c %64s + %-64s [", flag, eventsFound[mainEvent].name, eventsFound[pairEvent].name);
                calculateSize(str, saveValues[0]);
                printf("%c%9s,", flag1, str);
                calculateSize(str, saveValues[1]);
                printf("%c%9s]\n", flag2, str);

                papi_errno = PAPI_cleanup_eventset(EventSet);
                if (papi_errno != PAPI_OK) {
                    test_fail(__FILE__, __LINE__, "PAPI_cleanup_eventset failed.",
                              papi_errno);
                }

                papi_errno = PAPI_destroy_eventset(&EventSet);
                if (papi_errno != PAPI_OK) {
                    test_fail(__FILE__, __LINE__, "PAPI_destroy_eventset failed.",
                              papi_errno);
                }
            }
        }

        if (type == 0) {
            if (goodOnSame == 0) {
                printf("NO valid pairings of above events if both on the SAME device.\n");
            } else {
                printf("%i valid pairings of above events if both on the SAME device.\n", goodOnSame);
            }

            printf("%i unique pairings on SAME device were rejected as bad combinations.\n", badSameCombo);

            if (pairProblems > 0) {
                printf("%i pairings resulted in a change of one or both event values > 10%%.\n", pairProblems);
            } else {
                printf("No significant change in event values read for any pairings.\n");
            }
        } else {
            if (failOnDiff == 0) {
                printf("NO failed pairings of above events if each on a DIFFERENT device.\n");
            } else {
                printf("%i failed pairings of above events with each on a DIFFERENT device.\n", failOnDiff);
            }
        }
    }

    PAPI_shutdown();
    return(0);
}
