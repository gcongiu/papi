#include "hip/hip_runtime.h"
/****************************/
/* THIS IS OPEN SOURCE CODE */
/****************************/

/**
 * @file    hello_cupti.cu
 * @author  Heike Jagode
 *          jagode@eecs.utk.edu
 * Mods:    Giuseppe Congiu
 *          gcongiu@icl.utk.edu
 * test case for Example component
 *
 *
 * @brief
 *  This file is a very simple HelloWorld C example which serves (together
 *  with its Makefile) as a guideline on how to add tests to components.
 *  The papi configure and papi Makefile will take care of the compilation
 *  of the component tests (if all tests are added to a directory named
 *  'tests' in the specific component dir).
 *  See components/README for more details.
 *
 *  The string "Hello Cupti!" is mangled and then restored.
 */

#include <stdio.h>
#include "papi.h"

#define NUM_EVENTS 1

// Prototypes
__global__ void helloCupti(char*);

// Host function
int main(int argc, char** argv)
{
    int retval, i;
    int quiet;

    char str[] = "Hello Cupti!";

    int j;
    for (j = 0; j < 12; j++) {
        /* mangle contents of output */
        str[j] -= j;
    }

    hipSetDevice(0);

    char *d_str;
    size_t size = sizeof(str);
    hipMalloc((void**)&d_str, size);
    hipMemcpy(d_str, str, size, hipMemcpyHostToDevice);

    dim3 dimGrid(2);  // one block per word
    dim3 dimBlock(6); // one thread per character

    helloCupti<<<dimGrid, dimBlock>>>(d_str);

    hipDeviceSynchronize();
    hipMemcpy(str, d_str, size, hipMemcpyDeviceToHost);
    hipFree(d_str);

    return 0;
}

__global__ void
helloCupti(char* str)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    str[idx] += idx;
}
